#include "hip/hip_runtime.h"
#include "mbrot.h"

/*

    Program to generate images of the mandelbrot set.

    By Felipe Noronha and Rafael Tsuha at IME-USP.

*/

/* MACROS */
#define MAX 100 /* Max number of iterations */

/* Error handle */
#define ERRO(...) { \
        fprintf(stderr, __VA_ARGS__); \
        exit(EXIT_FAILURE); \
}

/* CUDA Error handle */
#define cudaERRO(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            ERRO(msg); \
        } \
    } while (0)

/* 'inline' makes the code be 'rewritten' in the call location, making it faster */
/* Transforms an interger value into a RGB color format */
inline void to_RGB(png_byte *ptr, int val) {

    /* Nomalize value */
	float color = ((float)val / MAX) * 255;
	ptr[0] = (int)color;
	ptr[1] = (int)color;
	ptr[2] = (int)color;
}

/* Parse data from argv[] to struct */
static struct aux_args* parse_args(const char* argv[]) {

    /* Static keyword will keep data until end of exec */
    static struct aux_args ret;

    ret.c0_real = strtof (argv[1], NULL);
    ret.c0_imag = strtof (argv[2], NULL);
    ret.c1_real = strtof (argv[3], NULL);
    ret.c1_imag = strtof (argv[4], NULL);
    ret.w = strtol (argv[5], NULL, 10);
    ret.h = strtol (argv[6], NULL, 10);
    ret.is_cpu = (strcmp(argv[7], "cpu") == 0);
    ret.threads = strtol (argv[8], NULL, 10);
    ret.saida = argv[9];

    return &ret;
}

/* Create a png image from a int array calc[], what is an 1D
   representation of a 2D array, where each position is a certain
   number of iterations */
void generate_image(struct aux_args *arg, int *calc) {

    FILE *file_ptr = NULL;
    png_structp png_ptr = NULL;
	png_infop info_ptr = NULL;
	png_bytep row = NULL;

    /* Opening new file for writing */
    file_ptr = fopen(arg->saida, "wb");

    if (file_ptr == NULL)
        ERRO("Could not open new file for writing.\n");

    /* Initialize new empty structure for png */
    png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);

    if (png_ptr == NULL)
        ERRO("Could not allocate new struct for png.\n");

    /* Initialize information structure for png */
    info_ptr = png_create_info_struct(png_ptr);

    if (info_ptr == NULL)
        ERRO("Could not allocate information structure for png.\n");

    /* From now on any error will be handled here */
    if (setjmp(png_jmpbuf(png_ptr)))
        ERRO("Error during png creation.\n");

    /* Initialize png_ptr with new file */
    png_init_io(png_ptr, file_ptr);

    /* Header of png for 8-bit colour depth */
    png_set_IHDR(png_ptr,
        info_ptr,
        arg->w,
        arg->h,
        8,
        PNG_COLOR_TYPE_RGB,
        PNG_INTERLACE_NONE,
        PNG_COMPRESSION_TYPE_BASE,
        PNG_FILTER_TYPE_BASE);

    /* writing down generated information */
    png_write_info(png_ptr, info_ptr);

    /* Used to build image */
    row = (png_byte *)malloc(3 * arg->w * sizeof(png_byte));

    if (row == NULL)
        ERRO("Could not allocate memory for image building.\n");

    /* write image data, one row at a time */
    for (int y = 0; y < arg->h; y++) {
        for (int x = 0; x < arg->w; x++) {
            to_RGB(&row[x*3], calc[y*arg->w + x]);
        }
        png_write_row(png_ptr, row);
    }

    /* End writing */
    png_write_end(png_ptr, NULL);

    /* Cleaning up */
    fclose(file_ptr);
    png_free_data(png_ptr, info_ptr, PNG_FREE_ALL, -1);
    png_destroy_write_struct(&png_ptr, (png_infopp)NULL);
    free(row);

    return;
}

/* --- CPU --- */

/* Iterate using maldelbrot rule in each potion */
unsigned cpu_iterate (float c_real, float c_imag) {
	float z_real = 0.0;
	float z_imag = 0.0;
	unsigned iter = 0;

	while (cpu_squared_moduleZ(z_real, z_imag) <= 4.0 && iter < MAX ) {

        /* Calculating new position */
        float nz_r = z_real*z_real - z_imag*z_imag + c_real;
		float nz_i = 2.0*z_real*z_imag + c_imag;
        z_real = nz_r;
        z_imag = nz_i;
		++iter;
	}
	return iter;
}

/* Receive input arguments in arg, an array calc, of size WxH, put the number
   of iterations corresponding to each pixel of image */
void cpu_make_iterations (struct aux_args *arg, int *calc) {

    unsigned long long int size = arg->w * arg->h;
    /* Variation of each iteration */
	float dx = (arg->c1_real - arg->c0_real)/(float)arg->w;
	float dy = (arg->c1_imag - arg->c0_imag)/(float)arg->h;

    #pragma omp parallel for num_threads(arg->threads)
	for (unsigned long long int i = 0; i < size; ++i) {

        /* Calculating the complex value of each postion */
		float x = ((i % arg->w) * dx) + arg->c0_real;
		float y = ((i / arg->w) * dy) + arg->c0_imag;
		calc[i] = cpu_iterate (x, y);
	}
}

/* Auxiliar function for complex module */
inline float cpu_squared_moduleZ (float z_real, float z_imag) {
	return z_real*z_real + z_imag*z_imag;
}

/* --- GPU --- */

/* Kernel for the execution */
__global__
void gpu_make_iterations (struct aux_args *arg, int *calc) {

    unsigned long long int size = arg->w * arg->h;
    /* Variation of each iteration */
	float dx = (arg->c1_real - arg->c0_real)/(float)arg->w;
	float dy = (arg->c1_imag - arg->c0_imag)/(float)arg->h;

    /* Index of this thread */
    unsigned long long int t_index = (blockIdx.x * blockDim.x) + threadIdx.x;
    /* Position index this thread is taking care of */
    unsigned long long int p_index = t_index;

    while (p_index < size) {

        /* Calculating the complex value of each postion */
    	float x = ((p_index % arg->w) * dx) + arg->c0_real;
    	float y = ((p_index / arg->w) * dy) + arg->c0_imag;
    	calc[p_index] = gpu_iterate(x, y);

        /* Moving to next session */
        p_index += (blockDim.x*gridDim.x);

    }

    return;
}

/* Iterate using maldelbrot rule in each potion */
__device__
unsigned gpu_iterate (float c_real, float c_imag) {
	float z_real = 0.0;
	float z_imag = 0.0;
	unsigned iter = 0;

	while ( (z_real*z_real + z_imag*z_imag) <= 4.0 && iter < MAX ) {

        /* Calculating new position */
        float nz_r = z_real*z_real - z_imag*z_imag + c_real;
		float nz_i = 2.0*z_real*z_imag + c_imag;
        z_real = nz_r;
        z_imag = nz_i;
		++iter;
	}
	return iter;
}

int main(int argc, const char* argv[]) {

    /* GOOD POSITIONS TO USE */
    /* ./mbrot -2.0 1.5 1.0 -1.5 1000 1000 <...> */

    /* Getting parameters */
    struct aux_args *arg = parse_args(argv);

    /* Used for iteration calculus */
	int *calc = (int *)malloc(arg->w * arg->h * sizeof (int));

    if (calc == NULL)
        ERRO("Could not allocate malloc in main().\n");


    if (arg->is_cpu)
	    cpu_make_iterations (arg, calc);

    else {
        struct aux_args *d_arg;
        int *d_calc;

        hipSetDevice(0);

        hipMalloc((void **)&d_arg, sizeof(struct aux_args));
        hipMalloc((void **)&d_calc, sizeof(int) * arg->w * arg->h);
        cudaERRO("hipMalloc failure");

        hipMemcpy(d_arg, arg, sizeof(struct aux_args), hipMemcpyHostToDevice);
        cudaERRO("hipMemcpy H2D failure\n");

        /* <<< blocks , threads >>> */
        gpu_make_iterations<<<2496, arg->threads>>>(d_arg, d_calc);
        cudaERRO("Kernel launch failure\n");

        /* Wait for all blocks/threads finishes their work */
        hipDeviceSynchronize();
        cudaERRO("hipDeviceSynchronize failure\n");

        hipMemcpy(calc, d_calc, arg->w*arg->h*sizeof(int), hipMemcpyDeviceToHost);
        cudaERRO("hipMemcpy D2H failure\n");

        hipFree(d_calc);
        hipFree(d_arg);
    }

    generate_image (arg, calc);

    /* Let it go, let it go */
    free(calc);

    return 0;
}
